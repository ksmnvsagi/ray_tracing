#include "hip/hip_runtime.h"
#include "sphere.cuh"
__device__ sphere::sphere(const point3& center, float radius, material* mat):center(center), radius(radius), mat(mat) {

}

__device__ bool sphere::hit(const ray& r, float t_min, float t_max, hit_record& record) const {
    vec3 oc = center - r.orig();
    float a = r.dir().squared_length();
    float h = dot(r.dir(), oc);
    float c = oc.squared_length() - radius*radius;
    float discriminant = h*h - a*c;
    if (discriminant < 0) return false;
    float sqrtd = sqrt(discriminant);
    float root = (h-sqrtd)/a;
    if (root <= t_min || t_max <= root) {
        root = (h+sqrtd) / a;
        if (root <= t_min || t_max <= root) return false;
    }
    record.t = root;
    record.p = r.at(record.t);
    record.normal = (record.p - center) / radius;
    record.front_face = dot(r.dir(), record.normal) < 0;
    record.mat = mat;
    return true;
}