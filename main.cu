#include "hip/hip_runtime.h"
#include "config.cuh"
#include "camera.cuh"
#include "scenes.cuh"
#include "quad.cuh"

__global__ void render(camera cam, color* buff, bvh** node, hiprandState* rand_states) {
    for (int y = threadIdx.y + blockIdx.y * blockDim.y; y<cam.image_height; y+=blockDim.y*gridDim.y) {
        for (int x = threadIdx.x + blockIdx.x * blockDim.x; x<cam.image_width; x+=blockDim.x*gridDim.x) {
            int index = x + y*cam.image_width;
            hiprandState my_state = rand_states[index];
            point3 pixel = (cam).pixel00 + x*(cam).delta_x + y*(cam).delta_y;
            color final_color(0, 0, 0);
            for (int i=0; i<(cam).num_samples; i++) {
                vec3 ray_dir = pixel + 0.5f*(cam).delta_x * cudaRand(&my_state, -1, 1) +
                               0.5f*(cam).delta_y*cudaRand(&my_state, -1, 1) - (cam).center;
                ray r((cam).center,ray_dir);
                final_color+=(cam).ray_color(r, *node, &my_state);
            }
            buff[index] = final_color/(float)(cam).num_samples;
        }
    }
}

__global__ void create_world(int size, bvh** node, image* img, hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
//        global(size, node, rand_state);
//        checker_spheres(size, node, rand_state);
//        earth(size, world, img, rand_state);
//        quads(size, world, rand_state);
        empty_cornell(size, node, img, rand_state);
    }
}

__global__ void free_world(int size, bvh** node) {
    delete *node;
}

__global__ void create_rand(int WIDTH, int HEIGHT, hiprandState* rand_states) {
    for (int y = threadIdx.y + blockIdx.y * blockDim.y; y<HEIGHT; y+=blockDim.y*gridDim.y) {
        for (int x = threadIdx.x + blockIdx.x * blockDim.x; x<WIDTH; x+=blockDim.x*gridDim.x) {
            int index = x + y*WIDTH;
            hiprand_init(1984+index, 0, 0, &rand_states[index]);
        }
    }
}

int main() {
    // to handle recursion during BVH construction
    hipDeviceSetLimit(hipLimitStackSize, 8192);

    const unsigned int WIDTH = 600;
    const unsigned int HEIGHT = 600;
    const unsigned int WORLD_SIZE = 8;
    const float ASPECT_RATIO = 1.0f;
    const point3 lookat = point3(278, 278, 0);
    const point3 lookfrom = point3(278, 278, -800);
    const float vfov = 40.0f;
    const unsigned int SAMPLES = 100;
    const int NUM_PIXELS = WIDTH*HEIGHT;
    camera cam(ASPECT_RATIO, WIDTH, lookfrom, lookat, vfov, SAMPLES);

    // query device
    hipDeviceProp_t prop{};
    hipGetDeviceProperties(&prop, 0);
    int SMs = prop.multiProcessorCount;
    std::cout<<"SMs: "<<prop.multiProcessorCount<<"\nCompute capability: "<<prop.major<<'.'<<prop.minor<<'\n';

    // host and device color buffers
    color* host_buff = (color*)malloc(NUM_PIXELS*sizeof(color));
    color* dev_buff;
    cudaCheck(hipMalloc((void**)&dev_buff, NUM_PIXELS*sizeof(color)));
    hipEvent_t start;
    hipEvent_t stop;
    cudaCheck(hipEventCreate(&start));
    cudaCheck(hipEventCreate(&stop));

    dim3 blocks(2*SMs, 2*SMs);
    dim3 threads(32, 32);

    // hiprand init
    hiprandState* rand_states;
    cudaCheck(hipMalloc((void**)&rand_states, NUM_PIXELS*sizeof(hiprandState)));
    create_rand<<<blocks, threads>>>(WIDTH, HEIGHT, rand_states);

    // world creation (must be done on the GPU due to virtual functions)
    bvh** node;
    cudaCheck(hipMalloc((void**)&node, sizeof(bvh*)));
    image host_earth_texture("..\\earthmap.jpg");
    image* dev_earth_texture;
    cudaCheck(hipMalloc((void**)&dev_earth_texture, sizeof(image)));
    cudaCheck(hipMemcpy(dev_earth_texture, &host_earth_texture, sizeof(image), hipMemcpyHostToDevice));
    create_world<<<1,1>>>(WORLD_SIZE, node, dev_earth_texture, rand_states);

    // render
    cudaCheck(hipEventRecord(start));
    render<<<blocks, threads>>>(cam, dev_buff, node, rand_states);
    cudaCheck(hipEventRecord(stop));
    cudaCheck(hipEventSynchronize(stop));
    float elapsed;
    cudaCheck(hipEventElapsedTime(&elapsed, start, stop));
    std::cout<<"Elapsed time: "<<elapsed<<" ms\n";
    // copy memory back to CPU
    cudaCheck(hipMemcpy(host_buff, dev_buff, NUM_PIXELS*sizeof(color), hipMemcpyDeviceToHost));

    // output image
    std::ofstream output("../image.ppm");
    output << "P3\n" << WIDTH << ' ' << HEIGHT << "\n255\n";
    for (int i=0; i<NUM_PIXELS; i++) write_color(output, host_buff[i]);
    output.close();

    // cleanup
    free_world<<<1,1>>>(WORLD_SIZE, node);
    cudaCheck(hipGetLastError());
    cudaCheck(hipDeviceSynchronize());
    cudaCheck(hipFree(dev_buff));
    cudaCheck(hipFree(rand_states));
    cudaCheck(hipFree(node));
    cudaCheck(hipFree(dev_earth_texture));
    cudaCheck(hipEventDestroy(start));
    cudaCheck(hipEventDestroy(stop));
}