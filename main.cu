#include "hip/hip_runtime.h"
#include "config.cuh"
#include "camera.cuh"
#include "scenes.cuh"
#include "quad.cuh"

__global__ void render(camera cam, color* buff, bvh** node, hiprandState* rand_states) {
    for (int y = threadIdx.y + blockIdx.y * blockDim.y; y<cam.image_height; y+=blockDim.y*gridDim.y) {
        for (int x = threadIdx.x + blockIdx.x * blockDim.x; x<cam.image_width; x+=blockDim.x*gridDim.x) {
            int index = x + y*cam.image_width;
            hiprandState my_state = rand_states[index];
            point3 pixel = (cam).pixel00 + x*(cam).delta_x + y*(cam).delta_y;
            color final_color(0, 0, 0);
            for (int i=0; i<(cam).num_samples; i++) {
                vec3 ray_dir = pixel + 0.5f*(cam).delta_x * cudaRand(&my_state, -1, 1) +
                               0.5f*(cam).delta_y*cudaRand(&my_state, -1, 1) - (cam).center;
                ray r((cam).center,ray_dir);
                final_color+=(cam).ray_color(r, *node, &my_state);
            }
            buff[index] = final_color/(float)(cam).num_samples;
        }
    }
}

__global__ void create_world(int size, hittable_list** world, bvh** node, image* img, hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
//        global(size, world, node, rand_state);
//        checker_spheres(size, world, node, rand_state);
//        earth(size, world, node, img, rand_state);
//        quads(size, world, node, rand_state);
        empty_cornell(size, world, node, img, rand_state);
    }
}

__global__ void free_world(int size, hittable_list** world, bvh** node) {
    for(int i=0; i < size; i++) {
//        delete ((quad*)list[i])->mat;
//        delete list[i];
    }
    delete *world;
    delete *node;
}

__global__ void create_rand(int WIDTH, int HEIGHT, hiprandState* rand_states) {
    for (int y = threadIdx.y + blockIdx.y * blockDim.y; y<HEIGHT; y+=blockDim.y*gridDim.y) {
        for (int x = threadIdx.x + blockIdx.x * blockDim.x; x<WIDTH; x+=blockDim.x*gridDim.x) {
            int index = x + y*WIDTH;
            hiprand_init(1984+index, 0, 0, &rand_states[index]);
        }
    }
}

int main() {
    // to handle recursion during BVH construction
    size_t stack_size = 8192;
    hipDeviceSetLimit(hipLimitStackSize, stack_size);

    const unsigned int WIDTH = 600;
    const unsigned int HEIGHT = 600;
    const unsigned int WORLD_SIZE = 8;
    // query device
    hipDeviceProp_t prop{};
    hipGetDeviceProperties(&prop, 0);
    int SMs = prop.multiProcessorCount;
    std::cout<<"SMs: "<<prop.multiProcessorCount<<"/nCompute capability: "<<prop.major<<'.'<<prop.minor<<'\n';

    int num_pixels = WIDTH*HEIGHT;
    // host and device color buffers
    color* host_buff = (color*)malloc(num_pixels*sizeof(color));
    color* dev_buff;
    cudaCheck(hipMalloc((void**)&dev_buff, num_pixels*sizeof(color)));
    hipEvent_t start;
    hipEvent_t stop;
    cudaCheck(hipEventCreate(&start));
    cudaCheck(hipEventCreate(&stop));

    dim3 blocks(2*SMs, 2*SMs);
    dim3 threads(32, 32);

    // hiprand init
    hiprandState* rand_states;
    cudaCheck(hipMalloc((void**)&rand_states, num_pixels*sizeof(hiprandState)));
    create_rand<<<blocks, threads>>>(WIDTH, HEIGHT, rand_states);

    // world creation (must be done on the GPU due to virtual functions!)
    hittable_list** world;
    cudaCheck(hipMalloc((void**)&world, sizeof(hittable*)));
    bvh** node;
    cudaCheck(hipMalloc((void**)&node, sizeof(bvh*)));
    // camera
    camera cam(1.0f, WIDTH, point3(278, 278, -800), point3(278, 278, 0), 40, 100);
    image host_earth_texture("..\\earthmap.jpg");
    image* dev_earth_texture;
    cudaCheck(hipMalloc((void**)&dev_earth_texture, sizeof(image)));
    cudaCheck(hipMemcpy(dev_earth_texture, &host_earth_texture, sizeof(image), hipMemcpyHostToDevice));
    create_world<<<1,1>>>(WORLD_SIZE, world, node, dev_earth_texture, rand_states);

    // render
    cudaCheck(hipEventRecord(start));
    render<<<blocks, threads>>>(cam, dev_buff, node, rand_states);
    cudaCheck(hipEventRecord(stop));
    cudaCheck(hipEventSynchronize(stop));
    float elapsed;
    cudaCheck(hipEventElapsedTime(&elapsed, start, stop));
    std::cout<<"Elapsed time: "<<elapsed<<" ms\n";
    // copy memory back to CPU
    cudaCheck(hipMemcpy(host_buff, dev_buff, num_pixels*sizeof(color), hipMemcpyDeviceToHost));

    // output image
    std::ofstream output("../image.ppm");
    output << "P3\n" << WIDTH << ' ' << HEIGHT << "\n255\n";
    for (int i=0; i<num_pixels; i++) write_color(output, host_buff[i]);
    output.close();
    // cleanup
    free_world<<<1,1>>>(WORLD_SIZE, world, node);
    cudaCheck(hipGetLastError());
    cudaCheck(hipDeviceSynchronize());
    cudaCheck(hipFree(dev_buff));
    cudaCheck(hipFree(rand_states));
    cudaCheck(hipFree(world));
    cudaCheck(hipEventDestroy(start));
    cudaCheck(hipEventDestroy(stop));
}