#include "hip/hip_runtime.h"
#include "quad.cuh"
#include "hittable_list.cuh"

__device__ quad::quad(const point3& Q, const vec3& u, const vec3& v, material* mat): Q(Q), u(u), v(v), mat(mat) {
    vec3 n = cross(u, v);
    normal = unit(n);
    D = dot(normal, Q);
    w = n/dot(n,n);
    set_bounding_box();
}
__device__ void quad::set_bounding_box() {
    aabb quad_diag1 = aabb(Q, Q + u + v);
    aabb quad_diag2 = aabb(Q + u, Q + v);
    bbox = aabb(quad_diag1, quad_diag2);
}
__device__ aabb quad::bounding_box() const {
    return bbox;
}
__device__ bool quad::hit(const ray& r, float t_min, float t_max, hit_record& record) const {
    float denominator = dot(normal, r.dir());
    if (fabs(denominator) < FLT_MIN) return false; // parallel ray
    float t = (D - dot(normal, r.orig()))/denominator;
    if (t_min > t || t_max < t) return false; // ray already hit something else
    point3 intersection = r.at(t);
    vec3 q_to_intersect = intersection - Q;
    float alpha = dot(w, cross(q_to_intersect, v));
    float beta = dot(w, cross(u, q_to_intersect));

    if (!is_interior(alpha, beta, record)) return false;
    record.t = t;
    record.p = intersection;
    record.mat = mat;
    record.front_face = dot(r.dir(), normal) < 0;
    record.normal = record.front_face ? normal : -normal;
    return true;
}
__device__ bool quad::is_interior(float a, float b, hit_record& record) const {
    if ((0.f > a || 1.f < a) || (0.f > b || 1.f < b)) return false;

    record.u = a;
    record.v = b;
    return true;
}

__device__ hittable_list* create_box(const point3& a, const point3& b, material* mat) {
    hittable** list = (hittable**)malloc(6);
    hittable_list* sides = new hittable_list(list, 6);
    point3 min = point3(fmin(a.x(), b.x()), fmin(a.y(), b.y()), fmin(a.z(), b.z()));
    point3 max = point3(fmax(a.x(), b.x()), fmax(a.y(), b.y()), fmax(a.z(), b.z()));
    vec3 dx = vec3(max.x() - min.x(), 0, 0);
    vec3 dy = vec3(0, max.y() - min.y(), 0);
    vec3 dz = vec3(0, 0, max.z() - min.z());

    sides->add(new quad(point3(min.x(), min.y(), max.z()),  dx,  dy, mat)); // front
    sides->add(new quad(point3(max.x(), min.y(), max.z()), -dz,  dy, mat)); // right
    sides->add(new quad(point3(max.x(), min.y(), min.z()), -dx,  dy, mat)); // back
    sides->add(new quad(point3(min.x(), min.y(), min.z()),  dz,  dy, mat)); // left
    sides->add(new quad(point3(min.x(), max.y(), max.z()),  dx, -dz, mat)); // top
    sides->add(new quad(point3(min.x(), min.y(), min.z()),  dx,  dz, mat)); // bottom

    return sides;
}